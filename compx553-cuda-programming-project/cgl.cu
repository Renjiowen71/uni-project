
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <chrono>
#include <sys/time.h>
#include <ctime>


using std::cout; using std::endl;
using std::chrono::duration_cast;
using std::chrono::milliseconds;
using std::chrono::seconds;
using std::chrono::system_clock;


/**
* range: the maximum of rows and columns of the board to print 
*/
void print_board(bool *board, int board_size, int range){
   cout << endl;
   for (int row = 0; row < range && row < board_size; row++) {
     for(int col=0; col < range && col < board_size; col++){
           cout << board[col + row*board_size];

     }
     cout<<endl;
   }
    
}

/**
* initialize a board array of board_size*board_size on the host
*
*/
void init_board(bool* board, int board_size){
  //starts with a simple pattern at the left corner region (0,0) to (range, range)
  int range = 128;
  //use a fixed seed for the same board pattern 
  srand(1);
  //get system time in seconds and use it as the random seed for different board patterns       
  //auto sec_since_epoch = duration_cast<seconds>(system_clock::now().time_since_epoch()).count();
  //srand(sec_since_epoch);
  for (int row = 1; row < board_size -1; row++) {
     for(int col= 1; col < board_size -1; col++){
         if(row < range && col < range){
               board[col + row * board_size] = rand()%2;
               continue;                      
        }
       board[col + row * board_size] = 0;
     }
  }
}

/**
*Implemention of the CPU version
*Any live cell with fewer than two live neighbours dies, as if by underpopulation.
*Any live cell with two or three live neighbours lives on to the next generation.
*Any live cell with more than three live neighbours dies, as if by overpopulation.
*Any dead cell with exactly three live neighbours becomes a live cell, as if by reproduction.
*Note: you can ignore the cells on four edges by setting row or col = 1 and row or col < board_size-1
*/
void nextGeneration(bool* board, bool* next_board, int board_size){
     
  for (int row = 1; row < board_size -1; row++) {
    for(int col= 1; col < board_size -1; col++){
      //Calculate live neighbors      
      int live_neighbors = 0;
      int index = row * board_size + col;
      for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
          if (!(i == 0 && j == 0)) {
            if (board[(row + i) * board_size + (col + j)]) {
              live_neighbors++;
            }
          }
        }
      }

      //Conditions necessary for life
      next_board[index] = (board[index] ? (live_neighbors == 2 || live_neighbors == 3) : (live_neighbors == 3));
    }
  }
}
/**
* Implemention of the GPU version without using shared memory
*
*/
__global__ void nextGenerationGPU(bool* board, bool* next_board, int board_size){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
    
  //Calculate Row and Column  
  int row = index / board_size;
  int col = index % board_size;
  
  if (row >= 1 && row < board_size - 1 && col >= 1 && col < board_size - 1) {
    
    //Calculate live neighbors  
    int live_neighbors = board[(row - 1) * board_size + (col - 1)] 
      + board[(row - 1) * board_size + col] 
      + board[(row - 1) * board_size + (col + 1)] 
      + board[row * board_size + (col - 1)] 
      + board[row * board_size + (col + 1)] 
      + board[(row + 1) * board_size + (col - 1)] 
      + board[(row + 1) * board_size + col] 
      + board[(row + 1) * board_size + (col + 1)];

    //Life's condition 
    next_board[index] = (board[index] ? (live_neighbors == 2 || live_neighbors == 3) : (live_neighbors == 3));
    
  }
}

const int TILE_SIZE = 16; // Define the size of the tile/block
/**
* Implemention of the GPU version using shared memory   
*
*/
__global__ void nextGenerationGPUSharedMemory(bool* board, bool* next_board, int board_size){
  __shared__ bool shared_board[TILE_SIZE+2][TILE_SIZE+2];
    
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  
  int shared_row = threadIdx.y+1;
  int shared_col = threadIdx.x+1;
  
  if (row >= 1 && row < board_size - 1 && col >= 1 && col < board_size - 1) {

    shared_board[shared_row-1][shared_col-1] = board[(row-1) * board_size + (col-1)];

    if (threadIdx.x>=TILE_SIZE-2){
      shared_board[shared_row-1][shared_col+1] = board[(row-1) * board_size + (col+1)];
    }

    if (threadIdx.y>=TILE_SIZE-2){
      shared_board[shared_row+1][shared_col-1] = board[(row+1) * board_size + (col-1)];
    }
    
    if (threadIdx.y>=TILE_SIZE-2&&threadIdx.x>=TILE_SIZE-2){
      shared_board[shared_row+1][shared_col+1] = board[(row+1) * board_size + (col+1)];
    }

    __syncthreads();

    // Calculate next board using shared board
    int live_neighbors = shared_board[shared_row - 1][shared_col - 1] // Top-left
      + shared_board[shared_row - 1][shared_col]     // Top
      + shared_board[shared_row - 1][shared_col + 1] // Top-right
      + shared_board[shared_row][shared_col - 1]     // Left
      + shared_board[shared_row][shared_col + 1]     // Right
      + shared_board[shared_row + 1][shared_col - 1] // Bottom-left
      + shared_board[shared_row + 1][shared_col]     // Bottom
      + shared_board[shared_row + 1][shared_col + 1]; // Bottom-right

    //Life's condition 
    next_board[row * board_size + col] = (shared_board[shared_row][shared_col]  ? (live_neighbors == 2 || live_neighbors == 3) : (live_neighbors == 3));
  }
}

// Conway's Game of Life Test 
void init_block_board(bool* board, int board_size){
  board[board_size+1] = true;
  board[board_size+2] = true;
  board[board_size*2+1] = true;
  board[board_size*2+2] = true;
}
void init_blinker_board(bool* board, int board_size){
  board[board_size+2] = true;
  board[board_size*2+2] = true;
  board[board_size*3+2] = true;
}
void init_glider_board(bool* board, int board_size){
  board[board_size+1] = true;
  board[board_size+3] = true;
  board[board_size*2+2] = true;
  board[board_size*2+3] = true;
  board[board_size*3+2] = true;
}
void init_LWSS_board(bool* board, int board_size){
  board[board_size+3] = true;
  board[board_size+4] = true;
  board[board_size*2+2] = true;
  board[board_size*2+3] = true;
  board[board_size*2+4] = true;
  board[board_size*2+5] = true;
  board[board_size*3+2] = true;
  board[board_size*3+3] = true;
  board[board_size*3+5] = true;
  board[board_size*3+6] = true;
  board[board_size*4+4] = true;
  board[board_size*4+5] = true;
}

int main(void)
{
  int board_size = 32768; //2048,4096,8192,16384,32768
  int print_range = 12;
    
  //Initialize the Board
  dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
  dim3 numBlocks((board_size + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                         (board_size + threadsPerBlock.y - 1) / threadsPerBlock.y);
  // int thread_block = 64;
  // int num_blocks = (board_size * board_size + thread_block - 1) / thread_block;
  bool* pre_board = new bool[board_size * board_size];
  bool* next_board = new bool[board_size * board_size];
  bool* d_pre_board;
  bool* d_next_board;
  init_board(pre_board, board_size);
  //init_block_board(pre_board, board_size);
  //init_blinker_board(pre_board,board_size);
  //init_glider_board(pre_board,board_size);
  //init_LWSS_board(pre_board,board_size);

  print_board(pre_board, board_size, print_range);
   
  hipMalloc((void**)&d_pre_board, sizeof(bool) * board_size * board_size);
  hipMalloc((void**)&d_next_board, sizeof(bool) * board_size * board_size);

  for (int i=0; i<10;i++){
    //run at least ten generations and measure the elapsed time for each generation
    auto start = std::chrono::high_resolution_clock::now();
      
    //run one generation

    // nextGeneration(pre_board, next_board, board_size);

    // cudaMemcpy(d_pre_board, pre_board, sizeof(bool) * board_size * board_size, cudaMemcpyHostToDevice);
    // nextGenerationGPU<<<num_blocks, thread_block>>>(d_pre_board, d_next_board, board_size);
    // cudaDeviceSynchronize();
    // cudaMemcpy(next_board, d_next_board, sizeof(bool) * board_size * board_size, cudaMemcpyDeviceToHost);

    hipMemcpy(d_pre_board, pre_board, sizeof(bool) * board_size * board_size, hipMemcpyHostToDevice);
    nextGenerationGPUSharedMemory<<<numBlocks, threadsPerBlock>>>(d_pre_board, d_next_board, board_size);
    hipDeviceSynchronize();
    hipMemcpy(next_board, d_next_board, sizeof(bool) * board_size * board_size, hipMemcpyDeviceToHost);

    auto end = std::chrono::high_resolution_clock::now();
    auto milliseconds = std::chrono::duration_cast<std::chrono::milliseconds>(end-start);
    std::cout << "Generation " << i+1 << " elapsed time: " << milliseconds.count() << " milliseconds" << std::endl;

    print_board(next_board, board_size, print_range);

    std::swap(pre_board, next_board);
  }

  hipFree(d_pre_board);
  hipFree(d_next_board);
  return 0;
}

